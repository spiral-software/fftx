#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath> // Without this, abs is the wrong function!
#include <random>

#include "rconv_dims.h"

#include "RealConvolution.hpp"
#include "fftx3utilities.h"

template<int DIM>
void rconvDimension(fftx::rconv<DIM>& a_transformer,
                    int a_rounds,
                    int a_verbosity)
{
  if (!a_transformer.isDefined())
    {
      return;
    }

  std::cout << "***** test " << DIM << "D real convolution on "
            << a_transformer.inputSize() << std::endl;

  RealConvolution<DIM> fun(&a_transformer);
  TestRealConvolution<DIM>(fun, a_rounds, a_verbosity);
}

void rconvSize(fftx::point_t<3> a_size,
               int a_rounds,
               int a_verbosity)
{
  fftx::box_t<3> fulldomain(fftx::point_t<3>
                            ({{rconv_dims::offx+1,
                                  rconv_dims::offy+1,
                                  rconv_dims::offz+1}}),
                            fftx::point_t<3>
                            ({{rconv_dims::offx+a_size[0],
                                  rconv_dims::offy+a_size[1],
                                  rconv_dims::offz+a_size[2]}}));
  
  fftx::box_t<3> halfdomain(fftx::point_t<3>
                            ({{rconv_dims::offx+1,
                                  rconv_dims::offy+1,
                                  rconv_dims::offz+1}}),
                            fftx::point_t<3>
#if FFTX_COMPLEX_TRUNC_LAST
                            ({{rconv_dims::offx+a_size[0],
                                  rconv_dims::offy+a_size[1],
                                  rconv_dims::offz+a_size[2]/2+1}})
#else
                            ({{rconv_dims::offx+a_size[0]/2+1,
                                  rconv_dims::offy+a_size[1],
                                  rconv_dims::offz+a_size[2]}})
#endif
                            );
  fftx::rconv<3> tfm(a_size); // does initialization
  rconvDimension(tfm, a_rounds, a_verbosity);
}
  
int main(int argc, char* argv[])
{
  // { SHOW_CATEGORIES = 1, SHOW_SUBTESTS = 2, SHOW_ROUNDS = 3};
  printf("Usage:  %s [verbosity=0] [rounds=20]\n", argv[0]);
  printf("verbosity 0 for summary, 1 for categories, 2 for subtests, 3 for all iterations\n");
  int verbosity = 0;
  int rounds = 20;
  if (argc > 1)
    {
      verbosity = atoi(argv[1]);
      if (argc > 2)
        {
          rounds = atoi(argv[2]);
        }
    }
  printf("Running with verbosity %d, random %d rounds\n", verbosity, rounds);

  /*
    Set up random number generator.
  */
  std::random_device rd;
  generator = std::mt19937(rd());
  unifRealDist = std::uniform_real_distribution<double>(-0.5, 0.5);

  /*
    2-dimensional tests.
  */
  //    rconv2::init();
  //    rconvDimension(rconv2::transform, rconv_dims::domain2, rconv_dims::fdomain2,
  //                   rounds, verbosity);
  //    rconv2::destroy();
  
  /*
    3-dimensional tests.
  */

  // rconvSize(fftx::point_t<3>({{  48,  48,  48 }}), rounds, verbosity);

  fftx::point_t<3> *ents = fftx_rconv_QuerySizes ();

  for ( int ind = 0; ents[ind][0] != 0; ind++ )
    {
      rconvSize(ents[ind], rounds, verbosity);
    }
  
  // rconvSize(fftx::point_t<3>({{  48,  48,  48 }}), rounds, verbosity);
  
  // fftx::point_t<3> extents = rconv_dims::domain3.extents();
  // fftx::rconv<3> tfm(extents); // does initialization
  // rconvDimension(tfm, rconv_dims::domain3, rconv_dims::fdomain3,
  //                rounds, verbosity);

  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
