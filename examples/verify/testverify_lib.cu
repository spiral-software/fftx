#include "hip/hip_runtime.h"
#include <cmath> // Without this, abs returns zero!
#include <random>

#include "fftx_mddft_public.h"
#include "fftx_imddft_public.h"
#include "fftx_mdprdft_public.h"
#include "fftx_imdprdft_public.h"
// #include "fftx_rconv_public.h"

#include "mddft.fftx.precompile.hpp"
#include "imddft.fftx.precompile.hpp"
#include "mdprdft.fftx.precompile.hpp"
#include "imdprdft.fftx.precompile.hpp"
// #include "rconv.fftx.precompile.hpp"

#include "fftx3utilities.h"

#include "device_macros.h"
#include "VerifyTransform.hpp"

int main(int argc, char* argv[])
{
  // { SHOW_CATEGORIES = 1, SHOW_SUBTESTS = 2, SHOW_ROUNDS = 3};
  printf("Usage:  %s [verbosity=0] [rounds=20]\n", argv[0]);
  printf("verbosity 0 for summary, 1 for categories, 2 for subtests, 3 for rounds\n");
  int verbosity = 0;
  int rounds = 20;
  if (argc > 1)
    {
      verbosity = atoi(argv[1]);
      if (argc > 2)
        {
          rounds = atoi(argv[2]);
        }
    }
  printf("Running with verbosity %d, random %d rounds\n", verbosity, rounds);

  /*
    Set up random number generator.
  */
  std::random_device rd;
  generator = std::mt19937(rd());
  unifRealDist = std::uniform_real_distribution<double>(-0.5, 0.5);

    // last entry is { 0, 0, 0 }
  fftx::point_t<3> *ents = fftx_mddft_QuerySizes ();
  
  for ( int ind = 0; ents[ind][0] != 0; ind++ )
    {
      fftx::point_t<3> sz = ents[ind];

      {
        fftx::mddft<3> tfm(sz);
        if (tfm.isDefined())
          {
            TransformFunction<3, std::complex<double>, std::complex<double>>
              fun(&tfm, -1);
            VerifyTransform<3, std::complex<double>, std::complex<double>>
              (fun, rounds, verbosity);
          }
        // verifyTransform(tfm, -1, rounds, verbosity);
       }

      {
        fftx::imddft<3> tfm(sz);
        if (tfm.isDefined())
          {
            TransformFunction<3, std::complex<double>, std::complex<double>>
              fun(&tfm, 1);
            VerifyTransform<3, std::complex<double>, std::complex<double>>
              (fun, rounds, verbosity);
          }
        // verifyTransform(tfm, 1, rounds, verbosity);
       }

      {
        fftx::mdprdft<3> tfm(sz);
        if (tfm.isDefined())
          {
            TransformFunction<3, double, std::complex<double>>
              fun(&tfm, -1);
            VerifyTransform<3, double, std::complex<double>>
              (fun, rounds, verbosity);
          }
        // verifyTransform(tfm, -1, rounds, verbosity);
      }

      {
        fftx::imdprdft<3> tfm(sz);
        if (tfm.isDefined())
          {
            TransformFunction<3, std::complex<double>, double>
              fun(&tfm, 1);
            VerifyTransform<3, std::complex<double>, double>
              (fun, rounds, verbosity);
          }
        // verifyTransform(tfm, 1, rounds, verbosity);
      }
    }

  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
